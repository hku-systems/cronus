#include "hip/hip_runtime.h"

#include "cukernel.h"

#define N 100

__global__ void addKernel(int* c, int* a, int* b, int size) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size) {
		c[i] = a[i] + b[i];
	}
}

#include <stdio.h>

extern "C" int compute() {

	int *darra, *darrb, *darrc;
	int all_correct = 1;

	int *arra = (int*) malloc(sizeof(int) * (N * N));
	int *arrb = (int*) malloc(sizeof(int) * (N * N));
	int *arrc = (int*) malloc(sizeof(int) * (N * N));

	hipMalloc((void**)&darra, (N * N) * sizeof(int));
	hipMalloc((void**)&darrb, (N * N) * sizeof(int));
	hipMalloc((void**)&darrc, (N * N) * sizeof(int));

	for (int i = 0;i < (N * N);i++) {
		arra[i] = i;
		arrb[i] = i;
	}

	hipMemcpy(darra, arra, (N * N) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(darrb, arrb, (N * N) * sizeof(int), hipMemcpyHostToDevice);

	addKernel<<<N, N>>>(darrc, darra, darrb, N * N);

	hipMemcpy(arrc, darrc, (N * N)*sizeof(int), hipMemcpyDeviceToHost);


	for (int i = 0;i < (N * N);i++) {
		if (arrc[i] != arra[i] + arrb[i]) {
			all_correct = 0;
			// printf("r[%d] is incorrect\n", i);
		}
	}

	if (all_correct) {
		fprintf(stderr, "all r[i] is correct\n");
	} else {
		fprintf(stderr, "some r[i] are incorrect\n");
	}

	hipFree(darra);
	hipFree(darrb);
	hipFree(darrc);

	return all_correct;
}
