#include <stdint.h>
#include <hip/hip_runtime.h>

__global__
void loop(uint32_t *data, uint32_t size, uint32_t n)
{
    int i;
//	for (i = 0; i < n/40; i++) {
	for (i = 0; i < n/5; i++) {
		if (i * 4 < size)
			data[i] = i + n;
    }
}
