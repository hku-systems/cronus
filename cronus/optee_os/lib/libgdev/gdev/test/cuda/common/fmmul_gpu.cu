#include <stdint.h>
#include <hip/hip_runtime.h>
__global__
void mul(float *a, float *b, float *c, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < n && j < n) {
        int idx = i * n + j;
        c[idx] = a[idx] * b[idx];
    }
}
