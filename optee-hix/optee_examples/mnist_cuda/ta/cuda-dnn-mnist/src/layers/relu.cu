#include "hip/hip_runtime.h"
#include "relu.cuh"

__global__
void kReLu(float *A, int aX, int aY, float* B) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < aX && y < aY) {
        if (A[y*aX + x] < 0.0) {
            B[y*aX + x] = 0;
        } else {
            B[y*aX + x] = A[y*aX + x];
        }
    }
}

ReLuLayer::ReLuLayer(int inputOutput) {
    this->input = this->output = inputOutput;
    this->weights = NULL;
    this->bias = NULL;
    this->deltaWeights = NULL;
    this->deltaBias = NULL;

    // Prepare output for forward and backprop
    this->outputForward = NULL;
    this->outputBackward = NULL;
}

Tensor2D* ReLuLayer::forward(Tensor2D* data) {
    this->inputData = data;

    if (!this->outputForward) {
        this->outputForward = new Tensor2D(data->getSize(X), data->getSize(Y));
    }

    dim3 threadsPerBlock(Configuration::reLuBlockSize, Configuration::reLuBlockSize);
    dim3 numBlocks((data->getSize(X) + threadsPerBlock.x)/threadsPerBlock.x,
                   (data->getSize(Y) + threadsPerBlock.y)/threadsPerBlock.y);
    kReLu<<<numBlocks, threadsPerBlock>>>(
        data->getDeviceData(), data->getSize(X), data->getSize(Y),
        this->outputForward->getDeviceData()
    );
    return this->outputForward;
}
 
Tensor2D* ReLuLayer::backward(Tensor2D* gradients) {
    dim3 threadsPerBlock(Configuration::reLuBlockSize, Configuration::reLuBlockSize);
    dim3 numBlocks((gradients->getSize(X) + threadsPerBlock.x)/threadsPerBlock.x,
                   (gradients->getSize(Y) + threadsPerBlock.y)/threadsPerBlock.y);
    kReLu<<<numBlocks, threadsPerBlock>>>(
        gradients->getDeviceData(), gradients->getSize(X), gradients->getSize(Y),
        gradients->getDeviceData()
    );
    return new Tensor2D(gradients->getSize(X), gradients->getSize(Y), gradients->getDeviceData());
}
