#include "hip/hip_runtime.h"
#include "crossentropy.cuh"

#define VERY_SMALL_NUMBER 1e-10

__global__
void kSoftMaxCrossEntropy(float *output, int oX, int oY, float* labels, float* y) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < oY) {
        // Calculate sum of exponents for whole column
        float sum = 0.0;
        for (int i = 0; i < oX; i++) {
            sum += exp(output[row*oX + i]);
        }
        if (abs(sum) < VERY_SMALL_NUMBER) {
            sum = VERY_SMALL_NUMBER;
        }

        // Softmax = exp(value) / sum(exp(allValues))
        // Subtract truth (which is one hot)
        for (int i = 0; i < oX; i++) {
            y[row*oX + i] = (exp(output[row*oX + i]) / sum) - labels[row*oX + i];
        }
    }
}

__global__
void kSoftMaxCrossEntropyLoss(float *output, int oX, int oY, float* labels, float* error) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < oY) {
        // Calculate sum of exponents for whole column
        float sum = 0.0;
        for (int i = 0; i < oX; i++) {
            sum += exp(output[row*oX + i]);
        }
        if (abs(sum) < VERY_SMALL_NUMBER) {
            sum = VERY_SMALL_NUMBER;
        }

        // Error = target * log(softmaxOutput) + (1 - target) * log(1 - softmaxOutput)
        float tmpError = 0.0;
        for (int i = 0; i < oX; i++) {
            float softmaxOutput = exp(output[row*oX + i]) / sum;
            tmpError -= labels[row*oX + i] * log(softmaxOutput) + 
                        (1 - labels[row*oX + i]) * log(1 - softmaxOutput);
        }
        atomicAdd(error, tmpError);
    }
}

__global__
void kSoftMaxCrossEntropyAccuracy(float *output, int oX, int oY, float* labels, float* accuracy) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < oY) {
        int maxIdx = 0;
        float maxValue = output[row*oX];
        for (int x = 1; x < oX; x++) {
            if (output[row*oX + x] > maxValue) {
                maxIdx = x;
                maxValue = output[row*oX + x];
            }
        }
        if (output[row*oX + maxIdx] > 1.0 - VERY_SMALL_NUMBER) {
            atomicAdd(accuracy, 1);
        }
    }
}

CrossEntropyLoss::CrossEntropyLoss() {}

float CrossEntropyLoss::getLoss(Tensor2D* networkOutput, Tensor2D* labels) {
    float error = 0.0;
    float* dError;
    hipMalloc((void**)&dError, sizeof(float));
    hipMemcpy(dError, &error, sizeof(float), hipMemcpyHostToDevice);

    dim3 threadsPerBlock = Configuration::crossEntropyGetMetricBlockSize;
    dim3 numBlocks((networkOutput->getSize(Y) + threadsPerBlock.x)/threadsPerBlock.x);
    kSoftMaxCrossEntropyLoss<<<numBlocks, threadsPerBlock>>>(networkOutput->getDeviceData(), networkOutput->getSize(X), networkOutput->getSize(Y), labels->getDeviceData(), dError);
    hipMemcpy(&error, dError, sizeof(float), hipMemcpyDeviceToHost);
    hipFree(dError);
    return error / networkOutput->getSize(Y);
}

float CrossEntropyLoss::getAccuracy(Tensor2D* networkOutput, Tensor2D* labels) {
    float accuracy = 0.0;
    float* dAccuracy;
    hipMalloc((void**)&dAccuracy, sizeof(float));
    hipMemcpy(dAccuracy, &accuracy, sizeof(float), hipMemcpyHostToDevice);

    dim3 threadsPerBlock = Configuration::crossEntropyGetMetricBlockSize;
    dim3 numBlocks((networkOutput->getSize(Y) + threadsPerBlock.x)/threadsPerBlock.x);
    kSoftMaxCrossEntropyAccuracy<<<numBlocks, threadsPerBlock>>>(networkOutput->getDeviceData(), networkOutput->getSize(X), networkOutput->getSize(Y), labels->getDeviceData(), dAccuracy);
    hipMemcpy(&accuracy, dAccuracy, sizeof(float), hipMemcpyDeviceToHost);
    hipFree(dAccuracy);
    return 100.0 * accuracy / networkOutput->getSize(Y);
}

Tensor2D* CrossEntropyLoss::calculate(Tensor2D* networkOutput, Tensor2D* labels, Tensor2D* output) {
    dim3 threadsPerBlock = Configuration::crossEntropyCalculateBlockSize;
    dim3 numBlocks((networkOutput->getSize(Y) + threadsPerBlock.x)/threadsPerBlock.x);
    kSoftMaxCrossEntropy<<<numBlocks, threadsPerBlock>>>(networkOutput->getDeviceData(), networkOutput->getSize(X), networkOutput->getSize(Y), labels->getDeviceData(), output->getDeviceData());
    return output;
}
